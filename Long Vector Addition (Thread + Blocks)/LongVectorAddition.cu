#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	//tid = thread index
	c[tid] = a[tid] + b[tid];
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

void random_ints(int *a, int M);

int main(void) {
	int *a, *b, *c; //pointers to host copies of the values
	int *d_a, *d_b, *d_c; //device copies of a,b,c
	int size = N * sizeof(int);

	//Alocating space in the device
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	//Alloc space of rhost copies of a,b,c
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	//Copy to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	//Launch function on GPU
	add << <(N+127)/128, 128 >> >(d_a, d_b, d_c);

	//Copy results back
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	int i;
	for (i = 0; i < N; i++) {
		if (i % 100000) {
			printf("%d + %d = %d\n", a[i], b[i], c[i]);
		}
	}

	//cleanup

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);


	return 0;
}

void random_ints(int *a, int M) {
	int i;

	for (i = 0; i < M; i++) {
		a[i] = rand() % 5000;

	}

}