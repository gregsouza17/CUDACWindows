#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../common/book.h"
#include "../common/cpu_anim.h"


//Global
#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f 

//setting texture references memory in GPU
texture<float> texConstSrc;
texture<float> texIn;
texture<float> texOut;

//Global

//KKKKKKKKKKKKKKKKKKKKKKKKKKKKK
__global__ void copy_const_kernel(float *iptr) {
	/*Given a grid to input temperatures iptr, copies the constant temperatures in the texture texConstSrc
	to the pointer iptr.*/

	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int offset = x + y *blockDim.x*gridDim.x;
	//Giving x,y unique values and linearizing it in offset

	//The constants are stored in the texConstSrc texture
	float c = tex1Dfetch(texConstSrc, offset);

	if (c != 0) iptr[offset] = c;


}

__global__ void blend_kernel(float *dst, bool dstOut) {
	/*Using texture memory we cant use directly the pointers to the memory, we will use a boolean
	flag instead to know which memory to use as input or output, sending the result through dst*/

	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int offset = x + y *blockDim.x*gridDim.x;


	//setting neighboorhod
	int left = offset - 1;
	int right = offset + 1;
	if (x == 0) left++;
	if (x == DIM - 1) right--;

	int top = offset - DIM;
	int bottom = offset + DIM;
	if (y == 0) top += DIM;
	if (y == DIM - 1) bottom -= DIM;

	float t,l,c,r,b;
	if (dstOut) { //if dstOut is TRUE
		t = tex1Dfetch(texIn,top);
		l = tex1Dfetch(texIn, left);
		c = tex1Dfetch(texIn, offset);
		r = tex1Dfetch(texIn, right);
		b = tex1Dfetch(texIn, bottom);
	} //We take the input from the texture texIn
	else { //ELSE
		t = tex1Dfetch(texOut, top);
		l = tex1Dfetch(texOut, left);
		c = tex1Dfetch(texOut, offset);
		r = tex1Dfetch(texOut, right);
		b = tex1Dfetch(texOut, bottom);
	} //We Take the input from texOut


	dst[offset] = c + SPEED*(t+b+r+l-4.0*c);

}
//KKKKKKKKKKKKKKKKKKKKKKKKKKKKKKKKKKKKKKKK

//AAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAA
struct DataBlock {
	//Setting bitmaps and grid of temperature
	unsigned char *output_bitmap;
	float *dev_inSrc; //Input
	float *dev_outSrc; //Output
	float *dev_constSrc; //Constant Sources
	CPUAnimBitmap	*bitmap;

	//Setting time variables to mesure time taken.
	hipEvent_t start, stop;
	float totalTime;
	float frames;
};

void anim_gpu(DataBlock *d, int ticks) {
	hipEventRecord(d->start, 0); //start recording time

								  //blocks and grids dimension
	dim3	blocks(DIM / 16, DIM / 16);
	dim3	threads(16, 16);
	//starting bitmap from the DataBlock
	CPUAnimBitmap	*bitmap = d->bitmap;

	//Make heat flow 90 times
	//since tex is global and bound we set up a boolean to flag which to use
	volatile bool dstOut = true;
	for (int i = 0; i < 90; i++) {
		//Setting input and output
		float *in , *out;
		if (dstOut) { //if True
			in = d->dev_inSrc;
			out = d->dev_outSrc;
		} //We take input from the expected
		else { //Else
			out = d->dev_inSrc;
			in = d->dev_outSrc;
		} //We change the order

		copy_const_kernel<<<blocks, threads>>>(in);
		blend_kernel<<<blocks,threads>>>(out , dstOut);
		dstOut = !dstOut;

	}
	//Make bitmap from output
	float_to_color << <blocks, threads >> >(d->output_bitmap, d->dev_inSrc);
	//copy bitmap to host
	hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost);

	//get and print time
	hipEventRecord(d->stop, 0);
	hipEventSynchronize(d->stop);
	float elapsedTime;

	hipEventElapsedTime(&elapsedTime, d->start, d->stop);

	d->totalTime += elapsedTime;
	++d->frames;

	printf("Avarage Time Per Frame %3.1f ms \n", d->totalTime / d->frames);

}


void anim_exit(DataBlock *d) {
	/*Cleans and wraps datablock*/

	hipUnbindTexture(texIn);
	hipUnbindTexture(texOut);
	hipUnbindTexture(texConstSrc);

	hipFree(d->dev_inSrc);
	hipFree(d->dev_outSrc);
	hipFree(d->dev_constSrc);

	hipEventDestroy(d->start);
	hipEventDestroy(d->stop);
}
//AAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAAA


int main(void) {
	//Setting Bitmaps
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;

	//Setting timeers = 0
	data.totalTime = 0;
	data.frames = 0;
	hipEventCreate(&data.start);
	hipEventCreate(&data.stop);



	//Allocating space to bitmap and sources.
	hipMalloc((void **)&data.output_bitmap, bitmap.image_size());

	hipMalloc((void **)&data.dev_inSrc, bitmap.image_size());
	hipMalloc((void **)&data.dev_outSrc, bitmap.image_size());
	hipMalloc((void **)&data.dev_constSrc, bitmap.image_size());

	//Bind texture reference to memory allocate in the device with the designed name (pg 126)
	hipBindTexture(NULL, texConstSrc, data.dev_constSrc, bitmap.image_size());
	hipBindTexture(NULL, texIn, data.dev_inSrc, bitmap.image_size());
	hipBindTexture(NULL, texOut, data.dev_outSrc, bitmap.image_size());


	//setting heaters in temp
	float *temp = (float *)malloc(bitmap.image_size());
	int i = 0;
	for (i = 0; i < DIM*DIM; i++) {
		temp[i] = 0;
		int x = i%DIM;
		int y = i / DIM;

		if ((x > 300) && (x < 600) && (y > 310 && (y < 601)))
			temp[i] = MAX_TEMP;
	}

	temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[DIM * 700 + 100] = MIN_TEMP;
	temp[DIM * 300 + 300] = MIN_TEMP;
	temp[DIM * 200 + 700] = MIN_TEMP;

	for (int y = 800; y < 900; y++) {
		for (int x = 400; x < 500; x++) {
			temp[x + y*DIM] = MIN_TEMP;
		}
	}

	hipMemcpy(data.dev_constSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);

	for (int y = 800; y < DIM; y++) {
		for (int x = 0;x < 200; x++) {
			temp[x + y*DIM] = MAX_TEMP;
		}
	}

	hipMemcpy(data.dev_inSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);

	free(temp);

	bitmap.anim_and_exit((void(*)(void*, int))anim_gpu, (void(*)(void*))anim_exit);

	return 0;
}